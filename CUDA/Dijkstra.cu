#include "hip/hip_runtime.h"
#include <stdio.h>
#include <limits.h>
#include <queue>
#include <hip/hip_runtime.h>
#include <cmath>
#include <algorithm>
#include "Dijkstra.cuh"

#include <thread>

#include "GraphParse.h"
#include "GraphMatrix.h"

#define BLOCK_SIZE 1024

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

Result** cuda_DijkstraAPSP(const GraphMatrix& graph) {
    Result** results = new Result*[graph.GetSize()];
    queue<int> q;

    GraphMatrix dist = GraphMatrix(graph, INT_MAX);
    GraphMatrix prev = GraphMatrix(graph, -1);



    return results;
}

__global__ void dev_min(const int* arr, const int* idxs, int size, int* out_vals, int* out_idxs) {
    int tidx = threadIdx.x + blockIdx.x * blockDim.x; // how far into the array we index
    int split = size >> 1; // array is split into two
    // we will compare pairs from each half

    extern __shared__ int minvals[]; // shared in the block
    int* argmins = (int*)&minvals[blockDim.x]; // arrays are just next to eachother

    if (tidx > split) { return; }

    if (size == 2) { printf("idxs[0] = %d, idxs[1] = %d\n", idxs[0], idxs[1]); }
    int min = arr[tidx];
    int minid = tidx;
    int otherid = split + tidx;

    //printf("Comparing index %d with index %d\n", tidx, otherid);
    if (otherid < size && arr[otherid] < min) {
        //printf("Index %d was smaller! (arr[%d] = %d, arr[%d] = %d\n\n", otherid, minid, min, otherid, arr[otherid]);
        min = arr[otherid];
        minid = otherid;
    }

    minvals[threadIdx.x] = min; // highest sharing we can do here is block-wide
    argmins[threadIdx.x] = minid;

    // should have minimum between pairs in first and second half of array in each block's
    // work set

    // now need to find minimum of all these

    // so lets the find the min within each block, since we are shared here
    // keep splitting, like we did for the full array

    for (int bsplit = (size < blockDim.x ? size : blockDim.x) >> 1; bsplit > 0; bsplit >>= 1) {
        if (threadIdx.x > bsplit) { return; } // dump any threads right of the split
        otherid = bsplit + threadIdx.x;
        if (size == 2) { printf("Reached here: otherid = %d, blockIdx.x = %d, blockDim.x = %d, bsplit = %d\n",
            otherid, blockIdx.x, blockDim.x, bsplit); }

        if ((otherid + blockIdx.x * blockDim.x) * 2 > size) { return; }

        __syncthreads();

        if (otherid < blockDim.x && minvals[otherid] < min) {
            min = minvals[otherid];
            minid = argmins[otherid];
        }
        if (blockIdx.x == 1 && min == 0 && bsplit == 512) {
            printf("tid = %d, otherid = %d, oidx = %d\n", threadIdx.x, otherid,
                otherid + blockIdx.x * blockDim.x);
        }
        minvals[threadIdx.x] = min;
        argmins[threadIdx.x] = minid;
        //if (blockIdx.x == 1) {
        //    printf("min for tid %d: %d ( bsplit = %d ) ( tidx = %d ) \n", threadIdx.x, min, bsplit, tidx);
        //    printf("minid for tid %d: %d ( bsplit = %d ) ( tidx = %d ) \n", threadIdx.x, minid, bsplit, tidx);
        //}
        //if (blockIdx.x == 1) {
        //    printf("minvals[%d]: %d\n", threadIdx.x, minvals[threadIdx.x]);
        //    printf("argmins[%d]: %d\n", threadIdx.x, argmins[threadIdx.x]);
        //}
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        //printf("Reached here\n");
        printf("minvals[0]: %d\n", minvals[0]);
        printf("argmins[0]: %d\n", argmins[0]);
        out_vals[blockIdx.x] = minvals[0];
        if (*idxs == -1) {
            out_idxs[blockIdx.x] = argmins[0];
            //printf("\nout_idxs[%d] = %d\n", blockIdx.x, out_idxs[blockIdx.x]);
        }
        else {
            //printf("argmins[0] = %d\n", argmins[0]);
            out_idxs[blockIdx.x] = idxs[argmins[0]];
        }
    }
}

/*__global__ void minArgMin(int* vals, int* idxs, int size, int* out_vals, int* out_idxs) {
    int tidx = threadIdx.x + blockIdx.x * blockDim.x;
    int split = size >> 1;

    extern __shared__ int minvals[];
    int* argmins = (int*)&minvals[blockDim.x];

    if (tidx > split) { return; }

    int min = vals[tidx];
    int minid = tidx;
    int otherid = split + tidx;

    if (otherid < size && vals[otherid] < min) {
        min = vals[otherid];
        minid = otherid;
    }

    minvals[threadIdx.x] = min; // highest sharing we can do here is block-wide
    argmins[threadIdx.x] = minid;

    for (int bsplit = blockDim.x >> 1; bsplit > 0; bsplit >>= 1) {
        if (threadIdx.x > bsplit) { return; } // dump any threads right of the split
        otherid = bsplit + threadIdx.x;
        if ((otherid + blockIdx.x * blockDim.x) * 2 > size) { return; }
        __syncthreads();

        if (otherid < blockDim.x && minvals[otherid] < min) {
            min = minvals[otherid];
            minid = argmins[otherid];
        }
        if (blockIdx.x == 1 && min == 0 && bsplit == 512) {
            printf("tid = %d, otherid = %d, oidx = %d\n", threadIdx.x, otherid,
                otherid + blockIdx.x * blockDim.x);
        }
        minvals[threadIdx.x] = min;
        argmins[threadIdx.x] = minid;
    }
}*/

int fastmin(int* arr, int size) {
    int oldsize = size;
    int* d_arr;

    gpuErrchk(hipMalloc(&d_arr, size*sizeof(int)));
    gpuErrchk(hipMemcpy(d_arr, arr, size*sizeof(int), hipMemcpyHostToDevice));

    int* idxs; int t[1] = {-1};
    gpuErrchk(hipMalloc(&idxs, size*sizeof(int)));
    gpuErrchk(hipMemcpy(idxs, t, sizeof(int), hipMemcpyHostToDevice));

    while (size > 1) {
        int grid_size = ceil((size / (double) BLOCK_SIZE) / 2);
        int mem_size = BLOCK_SIZE * (sizeof(int) * 2);

        int* out_vals;
        gpuErrchk(hipMalloc(&out_vals, grid_size*sizeof(int)));
        int* out_idxs;
        gpuErrchk(hipMalloc(&out_idxs, grid_size*sizeof(int)));


        dev_min<<<grid_size, BLOCK_SIZE, mem_size>>>(d_arr, idxs, size, out_vals, out_idxs);
        gpuErrchk(hipPeekAtLastError());
        gpuErrchk(hipDeviceSynchronize());

        size = grid_size;
        idxs = out_idxs;
        d_arr = out_vals;
    }


    printf("\n\n");

    int min; int argmin;
    gpuErrchk(hipMemcpy(&min, d_arr, sizeof(int), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(&argmin, idxs, sizeof(int), hipMemcpyDeviceToHost));

    printf("Min = %d at index %d\n", min, argmin);

    int* actualiter = min_element(arr, arr + oldsize);
    int actual = *actualiter; long int actualidx = actualiter - arr;

    printf("Actual min = %d at index %ld\n", actual, actualidx);

    return 0;
}

