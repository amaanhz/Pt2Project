#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "FWarsh.cuh"

#define BLOCK_SIZE 1024

using namespace std;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

Vec2::Vec2(int x, int y) : x(x), y(y) {}

Triple::Triple(Vec2 p1, Vec2 p2, Vec2 p3) : p1(p1), p2(p2), p3(p3) { }


__global__ void dep_block (int b, int num_blocks, int bl, int rem, int* dev_dist, int* dev_prev) {
    // B[b, b], B[b, b], B[b, b]

    if ( threadIdx.x >= bl || threadIdx.y >= bl ) return;
    int blockIndex = bl * b;
    int maxIndex = bl;
    if (b == num_blocks - 1) {
        maxIndex = rem;
        if (threadIdx.x > rem || threadIdx.y > rem) return;
    }

    int rowIndex = threadIdx.x * bl;
    int cell = rowIndex + threadIdx.y;

    // copy block data to shared memory
    extern __shared__ int dist[];
    int* prev = dist + bl;
    dist[cell] = dev_dist[cell]; prev[cell] = dev_prev[cell];

    for (int k = 0; k < maxIndex; k++) {
        int kRow = k * bl;
        if (dist[rowIndex + k] != INT_MAX && dist[kRow + threadIdx.y] != INT_MAX) {
            int t = dist[rowIndex + k] + dist[kRow + threadIdx.y];
            if (t < dist[cell]) {
                dist[cell] = t;
                prev[cell] = prev[kRow + threadIdx.y];
            }
        }
        __syncthreads(); // block must iterate in sync
    }

    // write-back
    dev_dist[cell] = dist[cell];
    dev_prev[cell] = prev[cell];
}

__global__ void pdep_blocks (int b, int num_blocks, int bl, int rem, int* dev_dist, int* dev_prev) {
    // B[b, i], B[b, b], B[b, i]
    // B[i, b], B[i, b], B[b, b]

    if ( threadIdx.x >= bl || threadIdx.y >= bl ) return;
    int blockIndex = bl * b;

    // find out which block we are


}

__global__ void indep_blocks (int b, int num_blocks, int bl, int rem, int* dev_dist, int* dev_prev) {
    // B[i, j], B[i, b], B[b, j]

    if ( threadIdx.x >= bl ) return;
}

Result** cuda_FWarsh(GraphMatrix& graph, int block_length) {
    int graphSize = graph.GetSize();
    int matSize = graphSize * graphSize;

    int* dev_dist; hipMalloc(&dev_dist, sizeof(int) * matSize);
    hipMemcpy(dev_dist, graph.GetMatrix(), sizeof(int) * matSize, hipMemcpyHostToDevice);

    int* dev_prev; hipMalloc(&dev_prev, sizeof(int) * matSize);
    GraphMatrix prev = GraphMatrix(graph, -1);
    for (int r = 0; r < graphSize; r++) {
        int rowIndex = r * graphSize;
        for (int c = 0; c < graphSize; c++) {
            if (graph[rowIndex + c] != INT_MAX) { prev[rowIndex + c] = r; } // set previous as in graph
        }
    }


    int num_blocks = (graph.GetSize() + block_length - 1) / block_length; // ceiling the value (1 axis)
    int rem = graph.GetSize() % block_length;
    if (rem == 0) { rem = block_length; }

    dim3 block_threads(block_length, block_length);

    int pdep_count = num_blocks * 2 - 2;
    dim3 pdep_dim(pdep_count / 2, pdep_count / 2);

    int indep_count = (num_blocks * num_blocks) - pdep_count - 1;
    dim3 indep_dim(indep_count / 2, indep_count / 2);

    size_t memsize = sizeof(int) * block_length * 2;

    for (int block = 0; block < num_blocks; block++) {
        dep_block<<<1, block_threads, memsize>>>(block, num_blocks, block_length, rem, dev_dist, dev_prev);
        pdep_blocks<<<pdep_dim, block_threads, memsize * 2>>>(block, num_blocks, block_length, rem, dev_dist, dev_prev);
        indep_blocks<<<indep_dim, block_threads, memsize * 2>>>(block, num_blocks, block_length, rem, dev_dist, dev_prev);
    }
}
